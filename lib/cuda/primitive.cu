
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// include <cutil.h>
// include <cuda.h>
#include <builtin_types.h>

#define INDX4(n, m) n + 4*m
#define INDX(a, b, c, d) a + b 

// device configuration

#define MULTIPROC_NR        4
#define MULTIPROC_MULTIPLE  2

#define BLOCK_SIZE   MULTIPROC_MULTIPLE*MULTIPROC_NR



/*

     Callable routines 

*/
extern "C" void primitive_gpu_(int*, int*, float* , float* , float* , float* , float* , float* );


extern "C" void test_int_(int* );
extern "C" void test_float_(float*);
extern "C" void test_ptr_(float*, int*);

/*

     Internal declarations

*/
__device__ void diffs(float*, float*, float*, float*, float*, float*, int, int);

/*

     This function performs the calculation

*/
__global__ void global_primitive(float* x, float* y, float* w,
                                 float* d, float* pr, float* pi,
                                 int Nx, int Nw) {

    // here the diffs must be summed in some sensible way

	diffs(x, y, w, d, pr, pi, Nx, Nw);


}



/*

    Solution of the 4x4 system of equations with Gauss elimination

*/
__device__ void gaussel(float* A, float* b, float* x) {

    int m;
    int n;

    __shared__ float a1[3][3];
    __shared__ float a2[2][2];
    __shared__ float b1[3];
    __shared__ float b2[2];

    for (m = 0; m < 3; m++) {
        for (n = 0; n < 3; n++) {
            a1[m][n] = A[INDX4(m,n)] - A[INDX4(m,0)]*A[INDX4(1,n)]/A[0];
        }
        b1[m] = b[m] - A[INDX4(m,0)]*b[0]/A[0];
    }
    
    for (m = 0; m < 2; m++) {
        for (n = 0; n < 2; n++) {
            a2[m][n] = a1[m][n] - a1[m][0]*a1[0][n]/a1[0][0];
        }
        b2[m] = b1[m] - a1[m][0]*b1[0]/a1[0][0];
    }

    x[3] = (a2[0][0]*b2[1] - a2[1][0]*b2[0])/(a2[0][0]*a2[1][1]-a2[1][0]*a2[0][1]);
    x[2] = (b2[0]/a2[0][0]) - (a2[0][1]/a2[0][0])*x[3];
    x[1] = (b1[0]/a1[0][0]) - (a1[0][1]/a1[0][0])*x[2] - (a1[0][2]/a1[0][0])*x[3];
    x[0] = (b[0] - A[INDX4(0,1)]*x[1] - A[INDX4(0,2)]*x[2] - A[INDX4(0,3)]*x[3])/A[0];

}


/*

    Subroutine for spline integration

*/
__device__ void get_abcd(float x1, float x2, float y1, float y2,
                         float d1, float d2, float* v) {

    __shared__ float M[16];
    __shared__ float b[4];

    M[0] = 1.0f;
    M[INDX4(1,0)] = 1.0f;
    M[INDX4(2,1)] = 1.0f;
    M[INDX4(3,1)] = 1.0f;
    M[INDX4(0,1)] = x1;
    M[INDX4(0,2)] = M[INDX4(0,1)]*x1;
    M[INDX4(0,3)] = M[INDX4(0,2)]*x1;
    M[INDX4(1,1)] = x2;
    M[INDX4(1,2)] = M[INDX4(1,1)]*x2;
    M[INDX4(1,3)] = M[INDX4(1,2)]*x2;
    M[INDX4(2,2)] = 2.0f*x1;
    M[INDX4(2,3)] = 3.0f*x1*x1;
    M[INDX4(3,2)] = 2.0f*x2;
    M[INDX4(3,3)] = 3.0f*x2*x2;

    b[0] = y1;
    b[1] = y2;
    b[2] = d1;
    b[3] = d2;

    gaussel(M, b, v);

} 


/*

    Multiplication of two complex numbers with and without constructor

*/
__device__ float2 m2c(float2 a, float2 b) {

    return make_float2(a.x*b.x - a.y*b.y,a.x*b.y + a.y*b.x);

}


/*

    Real by complex multiplication with and without constructor

*/
__device__ float2 mrc(float r, float2 a) {

    return make_float2(r*a.x,r*a.y);

}


/*

    Complex addition without constructor

*/
__device__ float2 d2c(float2 a, float2 b) {

    return make_float2(a.x - b.x,a.y - b.y);

}

/*

    Calculates integrals between interpolation points

*/
__device__ void diffs(float* x, float* y, float* w,
                      float* d, float* pr, float* pi,
                      int Nx, int Nw) {


    float x1, x2, y1, y2, d1, d2;
    float2 BeB, AeA, B2eB, A2eA, dI0, dI1, dI2, dI3;
    float2 I0, I1, I2, I3, eA, eB, iom; 
    float2 aux;
    
    __shared__ float A[4];

    int i;
    int k;

    // chose i and k according to the thread id

    i = 0;
    k = 0;

    // here I load from global to shared memory
    x1 = x[i];
    x2 = x[i+1];
    y1 = y[i];
    y2 = y[i+1];
    d1 = d[i];
    d2 = d[i+1];

    // calculation

    get_abcd(x1,x2,y1,y2,d1,d2,A);

    eA  = make_float2(cosf(w[k]*x1),sinf(w[k]*x1));
    eB  = make_float2(cosf(w[k]*x2),sinf(w[k]*x2));
    iom = make_float2(0.0f, -1.0f/w[k]);
 
    BeB = mrc(x2,eB);
    AeA = mrc(x1,eA);
    B2eB = mrc(x2,BeB);
    A2eA = mrc(x1,AeA);
    dI0 = make_float2(eB.x-eA.x,eB.y-eA.y);
    dI1 = make_float2(BeB.x-AeA.x,BeB.y-AeA.y);
    dI2 = make_float2(B2eB.x-A2eA.x,B2eB.y-A2eA.y);
    aux = mrc(x2,B2eB);
    dI3 = make_float2(x2*B2eB.x,x2*B2eB.y);
    dI3.x = dI3.x - aux.x;
    dI3.y = dI3.y - aux.y;

    I0 = m2c(iom,dI0);
    aux = d2c(dI1,I0);
    I1 = m2c(iom,aux);
    aux = mrc(2.0f,I1);
    aux = d2c(dI2,aux);
    I2 = m2c(iom,aux);
    aux = mrc(3.0f,I2);
    aux = d2c(dI2,aux);
    I3 = m2c(iom,aux);

    // returning to global memory

    pr[INDX(i+1,k,Nk,Nw)] = A[0]*I0.x + A[1]*I1.x + A[2]*I2.x + A[3]*I3.x;
    pi[INDX(i+1,k,Nk,Nw)] = A[0]*I0.y + A[1]*I1.y + A[2]*I2.y + A[3]*I3.y;

}


/*

   Function callable from host 

*/
void primitive_gpu_(int* Nxi, int* Nwi, float* x, float* y, float* w, float* d, float* pr, float* pi) {

   int nThread;
   int gridSize;
   int Nx, Nw, i;
   struct hipDeviceProp_t dprop;


   Nx = *Nxi;
   Nw = *Nwi;

   hipGetDeviceCount(&i);
   hipGetDeviceProperties(&dprop,0);
   

   printf("primitive_gpu ...%i - %i \n",i,dprop.multiProcessorCount);

   printf("   %i %i \n",Nx,Nw);

   /* allocate device memory */

   float* d_x;
   float* d_y;
   float* d_w;
   float* d_d;
   float* d_pr;
   float* d_pi;


   // number of threads and the size of the grid
   nThread = Nx*Nw; 
   gridSize = int(nThread/ int(BLOCK_SIZE));

   if (gridSize*BLOCK_SIZE != nThread) gridSize++;

   printf("   %i %i \n",Nx,Nw);
   printf("   block size = %i \n",BLOCK_SIZE);
   printf("   grid size = %i \n",gridSize);
   printf("   number of threads = %i \n", nThread);
   
   printf("%f \n",x[0]);
   printf("%f \n",x[1]);


   // copy into device memory
   hipMalloc((void**)&d_x,Nx);
   hipMemcpy(d_x,x,Nx,hipMemcpyHostToDevice);
   hipMalloc((void**)&d_y,Nx);
   hipMemcpy(d_y,y,Nx,hipMemcpyHostToDevice);
   hipMalloc((void**)&d_d,Nx);
   hipMemcpy(d_d,d,Nx,hipMemcpyHostToDevice);
   hipMalloc((void**)&d_pr,Nx*Nx);
   hipMalloc((void**)&d_pi,Nx*Nx);
   hipMalloc((void**)&d_w,Nw);
   hipMemcpy(d_w,w,Nw,hipMemcpyHostToDevice);


   // call GPU
   dim3 dimBlock(BLOCK_SIZE);
   dim3 dimGrid(gridSize);
   global_primitive<<<dimBlock, dimGrid>>>(d_x, d_y, d_w, d_d,
                                           d_pr, d_pi, Nx, Nw);


   hipMemcpy(pi, d_pi,Nx*Nw,hipMemcpyDeviceToHost);
   hipMemcpy(pr, d_pr,Nx*Nw,hipMemcpyDeviceToHost);

   // copy data to the host

   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_d);
   hipFree(d_w);
   hipFree(d_pr);
   hipFree(d_pi);

   printf("... primitive_gpu finished\n");

}

void test_int_(int* N) {

  int Ni;

  Ni = *N;

  printf("%i \n", Ni);
  
  *N = 2;

}

void test_float_(float* r) {

  printf("%f \n",r[0]);

}

void test_ptr_(float* x, int* N) {
	
	int Ni;
	Ni = *N;
	x[Ni-1] = 10.001;
	
}
