/*
	You can call cudaTest() from a C code. This function:
	returns 1 if cuda works well
		0 if no GPU is present
		-1 if an error in computation occured
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
const float t = 2.5;//multiplicative parameter
const int N= 1024;//matrixes are NxN
const int blockSize= 16;//blocks are 16x16


/*
	cuda kernel
	Each thread multiplies one element of the matrix by the t parameter
*/
__global__ void _multiplyArray(float* a, float t, int N){
    int x = blockIdx.x* blockDim.x+ threadIdx.x;
    int y = blockIdx.y* blockDim.y+ threadIdx.y;
    
    int i= x+ N*y;
    
    a[i]= t*a[i];
}

int cudaTest(){
    /*tests count of present cuda devices
    This doesn't work yet. I dont't know how to determine there is certainly the emulation mode in progress.*/
    int i;
    /*determines whether a cuda capable device is used*/
    hipGetDevice(&i);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    if ((prop.major< 1)||(prop.major> 1)) {/*in Feb 09 all cuda enabled devices have prop.major==1*/
    	hipGetDeviceCount(&i);
	if (i< 2) {/*no other device in the system*/
		return 0;/*no GPU*/
	} else {
		printf("You are not using the cuda capable device!");
		return 0;
	}
    }

    /*host array initialization*/
    float f[N*N];
    for (i=0;i<N*N;i++) f[i]= 1;


    const int size= N*N*sizeof(float);
    
    /*initializes the device array and fills it with the content of f*/
    float *_f;
    hipMalloc((void**) &_f, size);
    hipMemcpy(_f, f, size, hipMemcpyHostToDevice);
    
    /*executes the kernel*/
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);
    _multiplyArray<<<dimGrid, dimBlock>>>(_f, t, N);
    
    /*copies data back to the host array f and cleans the memory up*/
    hipMemcpy(f, _f, size, hipMemcpyDeviceToHost);
    hipFree(_f);
    
    /*tests the result*/
    for (i=0;i<N*N;i++) 
    	if (f[i]!= t) return -1; /*computation failed*/
    
    return 1; /*everything all right*/
}

