/*
	This file contains method nosecuda_polar_1(...) which returns response function S(t) in time. Theory can be found in the white paper Absorption Spectrum at NOSE web.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
	Computes response functions S(t) in time using CPU.
	Parameters: 
		int N[3]: N[1] = number of levels in the ground state band
		          N[2] = number of levels in the excited state band
			  N[3] = number of the time steps
		float rU[N[0]*N[1]*N[0]*N[1]*N[2]] = real part of the coherence evolution superoperator in time. The superoperator has 4 indeces, the 5th dimension represents discrete time.

		float iU[N[0]*N[1]*N[0]*N[1]*N[2]] = imaginary part of the coherence evolution superoperator. The superoperator has 4 indeces, the 5th dimension represents discrete time.
		float d[N[0]*N[1]] = transition dipole moment elements
		float rho0[N[0]] = diagonal elements of the equilibrium ground state density matrix
		float S[2*N[0]] = response function in time (both real and imaginary parts)
*/

float *nosecpu_polar_1(int* N, float* rU, float* iU, float* d, float* rho0){
	/*creates S array*/
	float *S;
	S= (float*)malloc(N[2]*2*sizeof(float));
	
	int t;
	int i, j, a, b;
	for (t=0;t<N[2];t++){ 
		//puts initial zeros into S
		S[2*t]= 0;
		S[2*t+1]= 0;

		for (i=0;i<N[0];i++){
			for (j=0;j<N[0];j++){
				for (a=0;a<N[1];a++){
					for (b=0;b<N[1];b++){
						/*In rU (resp. iU) the relation between rU[n] and indeces i, j, a, b, t is n= i+ N[0]*(a+ N[1]*(j+ N[0]*(b+ N[1]*t))). 
						In d array the relation is n=i+ N[0]* a.*/

						/*real part*/
						S[2*t]+= d[b*N[0]+j]* d[a*N[0]+i]* rU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]*rho0[i];
						/*imaginary part*/
						S[2*t+1]+= d[b*N[0]+j]* d[a*N[0]+i]* iU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]*rho0[i];
					}
				}
			}
		}
	}

	/*returns S(t)*/
	return S;
}
/*
int main(){//tests nosecuda_polar1 with simple parameters
	float *S;
	//initialization of some parameters / these parameters should make S[0]=1+0i
	int N[3]={2,2,1};
	float rU[16]={1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
	float iU[16]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	float d[4]={1,1,1,1};
	float rho0[2]={1,1};

	S= (float*)nosecuda_polar_1(N, rU, iU, d, rho0);//computes S
	int i;
	for (i=0;i<1;i++)//writes S array
		printf("S(%i)=%f+i%f\n", i, S[2*i], S[2*i+1]);
	free(S);
}*/
