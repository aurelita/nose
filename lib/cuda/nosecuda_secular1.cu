/*
	This file contains method nosecuda_secular_1(...) which returns response function S(t) in time. The secular approximation U_{bjai}(t)=\delta_ab \delta_ij U_{aiai}(t). Theory can be found in the white paper Absorption Spectrum at NOSE web.

	ATTENTION: This version is not yet optimized!

	At this state only N[0]==1 works as it should. 
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

int BLOCKDIM=4;//size of thread blocks
/*
	Computes response functions S(t) in time using GPU.
	Parameters: 
		int N[3]: N[1] = number of levels in the ground state band
		          N[2] = number of levels in the excited state band
			  N[3] = number of the time steps
		float rU[N[0]*N[1]*N[2]] = real part of the coherence evolution superoperator in secular approximation. The superoperator has 4 indeces, the 5th dimension represents discrete time. 1st and 3rd indeces and 2nd and 4th indeces are the same.

		float iU[N[0]*N[1]*N[2]] = imaginary part of the coherence evolution superoperator in secular approximation. The superoperator has 4 indeces, the 5th dimension represents discrete time.
		float d[N[0]*N[1]] = transition dipole moment elements
		float rho0[N[0]] = diagonal elements of the equilibrium ground state density matrix
		float S[2*N[0]] = response function in time (both real and imaginary parts)
*/

void checkCUDAError(const char* msg);

__global__ void _evolveS(int N0, int N1, int N2, float *data, float *rS, float *iS){ //rU, iU don't contain a!=b || i!=j elements yent
	int t= (gridDim.x*blockIdx.y+ blockIdx.x)*blockDim.x*blockDim.y+ blockDim.x*threadIdx.y+threadIdx.x;//calculates time of current thread
	//if (t>=N2) return;//because 4x4 blocks are alocated, some threads at the end are not necessary
	int bIdx= gridDim.x*blockIdx.y+blockIdx.x;
	int tIdx= blockDim.x*threadIdx.y+threadIdx.x;

	//initialized temporary variables
	float im= 0;
	float re= 0;
	//int BLOCKSIZE= blockDim.x*blockDim.y;
	int size= N0*N1*N2/(gridDim.x*gridDim.y);//size of rU (resp iU) region supplied to one block (in floats)
	int _rUOffset= bIdx*size;//offset of _rU according to first address of rU
	int _iUOffset= bIdx*size;//offset of _iU according to first address of iU

	//shared memory duplicates of global memory variables
	extern __shared__ float _sData[];//sData[];//all dynamically alocated arrays in shared memory start at the same address. So I have to create offsets. Structure of sData is: rU[size],iU[size],d[N0*N1],rho0[N0]
	float *_rU, *_iU, *_d, *_rho0;
	float *rU, *iU, *d, *rho0;
	rU= &data[_rUOffset];//1*100*16
	iU= &data[N0*N1*N2+ _iUOffset];
	d= &data[2*N0*N1*N2];
	rho0= &data[2*N0*N1*N2+ N0*N1];

	_rU= _sData;
	_iU= &_rU[size];
	_d= &_iU[size];
	_rho0= &_d[N0*N1];
	
	//copy rU, iU regions and d and rho0 into shared memory / threads should be coalesced
	int i, a;//indeces i, a 
	for (i=tIdx;i<size;i+= 16){
		_rU[i]= rU[i];
		_iU[i]= iU[i];
		if (i<N0*N1) {
			_d[i]= d[i];
			if (i<N0){
				_rho0[i]= rho0[i];	
			}
		}
	}

	__syncthreads();//all values copied into the shared memory (by all threads)

	//calculates S in secular approximation
	for (i=0;i<N0;i++){
		for (a=0;a<N1;a++){
			/*real part*/
			re+= _d[a*N0+i]* _d[a*N0+i]* _rU[i+N0*(a+N1*tIdx)]*_rho0[i];
			/*imaginary part*/
			im+= _d[a*N0+i]* _d[a*N0+i]* _iU[i+N0*(a+N1*tIdx)]*_rho0[i];
		}
	}
	__syncthreads();//all threads have accomplished their computation

	//stores resulsts into S / threads are coalesced because t=threadIdx
	rS[t]= re;
	iS[t]= im;
}

float *nosecuda_secular_1(int *N, float *rU, float *iU, float *d, float *rho0){
	//input data serialization
	float *data, *_data;//this array will by passed to the kernel. It containes serialized rU,iU,d,rho0
	int size= N[0]*N[1]*N[2];//size of rU array in floats
	data= (float*)malloc(sizeof(float)*(2*size+N[0]*N[1]+N[0]));
	int i, t;
	for (i=0;i<size;i++) data[i]= rU[i];
	for (i=0;i<size;i++) data[i+size]= iU[i];
	for (i=0;i<N[0]*N[1];i++) data[i+2*size]= d[i]; 
	for (i=0;i<N[0];i++) data[i+ 2*size+ N[0]*N[1]]= rho0[i];

	/*creates S array and arrays to store data from GPU*/
	float *S, *iS, *rS, *_iS, *_rS;
	S= (float*)malloc(2*N[2]*sizeof(float));
	iS= (float*)malloc(N[2]*sizeof(float)); 
	rS= (float*)malloc(N[2]*sizeof(float)); 

	/*initializes device variables*/
	/*allocate _data array*/
	hipMalloc((void**) &_data, (2*size+N[0]*N[1]+N[0])*sizeof(float)); 
	hipMemcpy(_data, data, (2*size+N[0]*N[1]+N[0])*sizeof(float), hipMemcpyHostToDevice);
	/*allocate S array*/
	hipMalloc((void**) &_iS, N[2]*sizeof(float));//N[2] contains real 
	hipMalloc((void**) &_rS, N[2]*sizeof(float));//N[2] contains real

	//calculates grid dimension
	dim3 dimBlock(BLOCKDIM, BLOCKDIM);

	int b= ceil(N[2]/(float)(BLOCKDIM*BLOCKDIM));//number of blocks needed
	i=1;//grid dimension in blocks
	while ((i+1)*(i+1)<=b) i+= 1;//find greatest square grid whose area is lesser than b
	dim3 dimGrid(i, i+ceil((b-i*i)/(float)i));

	//initialize testing f array
	/*float *_f, *f;
	cudaMalloc((void**) &_f, sizeof(float));*/

	/*kernel execution*/
	_evolveS<<<dimGrid, dimBlock, 16000>>>(N[0], N[1], N[2], _data, _rS, _iS);
	checkCUDAError("kernel invocation");

	/*copy result from the device to the host*/
	hipMemcpy(rS, _rS, N[2]*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(iS, _iS, N[2]*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("cpy");

	//copy data from rS and iS to S
	for (t=0;t<N[2];t++) {
		S[2*t]= rS[t];
		S[2*t+1]= iS[t];
	}
	/*f= (float*)malloc(sizeof(float));
	cudaMemcpy(f, _f, sizeof(float), cudaMemcpyDeviceToHost);
	printf("f=%f\n", *f);
	cudaFree(_f);*/
	

	/*free used memory*/
	hipFree(_data);
	hipFree(_iS);
	hipFree(_rS);
	free(data);
	free(rS);
	free(iS);
	
	/*returns S(t)*/
	return S;//the function which calls this method should free S
}

float random01(){//returns random number between 0 and 1
	return rand()/(float)RAND_MAX;
}

int main(int argc, char **argv){//tests nosecuda_secular1 function
	//set dimensios of parameters
	int N[3]={1, 64, 64};
	float *d, *rU, *iU, *rho0;
	d= (float*)malloc(N[1]*N[0]*sizeof(float));
	rho0= (float*)malloc(N[0]*sizeof(float));
	rU= (float*)malloc(N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float));
	iU= (float*)malloc(N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float));
	int i, j, a, b, t;

	//set parameters to random values
	for (i=0;i<N[0];i++){//set rho0, d
		rho0[i]= random01()*10.0;
		for (a=0;a<N[1];a++){
			d[i+N[0]*a]= random01()*10;//float between 0 and 100
		}
	}
	for (t=0;t<N[2];t++){//set rU, iU
		for (i=0;i<N[0];i++){//set d
			for (a=0;a<N[1];a++){
				for (j=0;j<N[0];j++){
					for (b=0;b<N[1];b++){
						rU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]= random01();
						iU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]= random01();
					}
				}
			}
		}
	}
	//perform cpu computation
	float cpuS[2*N[2]];
	for (t=0;t<N[2];t++){//set rU, iU
		cpuS[2*t]= 0;
		cpuS[2*t+1]= 0;
		for (i=0;i<N[0];i++){//set d
			for (a=0;a<N[1];a++){
				cpuS[2*t]+= d[i+a*N[0]]* d[i+a*N[0]]* rU[i+N[0]*(a+N[1]*t)]* rho0[i];
				cpuS[2*t+1]+= d[i+a*N[0]]* d[i+a*N[0]]* iU[i+N[0]*(a+N[1]*t)]* rho0[i];
			}
		}
	}
	//perform gpu computation
	float *S= nosecuda_secular_1(N, rU, iU, d, rho0);
	//compare both cpu and gpu methods and print results
	for (t=0;t<N[2];t++){
		printf("S[%i]=%f+i%f;cpu: %f+i%f\n", t, S[2*t], S[2*t+1], cpuS[2*t], cpuS[2*t+1]);
	}
	//free memory
	free(d);
	free(rho0);
	free(rU);
	free(iU);
	free(S);
}

void checkCUDAError(const char *msg){//checks for cudaErrors
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {   
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
//		exit(EXIT_FAILURE);
	}    
}

