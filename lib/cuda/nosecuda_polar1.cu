/*
	This file contains method nosecuda_polar_1(...) which returns response function S(t) in time. Theory can be found in the white paper Absorption Spectrum at NOSE web.

	ATTENTION: This version is not yet optimized!
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int BLOCKDIM=16;//size of thread blocks
/*
	Computes response functions S(t) in time using GPU.
	Parameters: 
		int N[3]: N[1] = number of lovels in the ground state band
		          N[2] = number of levels in the excited state band
			  N[3] = number of the time steps
		float rU[N[0]*N[1]*N[0]*N[1]*N[2]] = real part of the coherence evolution superoperator in time. The superoperator has 4 indeces, the 5th dimension represents discrete time.

		float iU[N[0]*N[1]*N[0]*N[1]*N[2]] = imaginary part of the coherence evolution superoperator. The superoperator has 4 indeces, the 5th dimension represents discrete time.
		float d[N[0]*N[1]] = transition dipole moment elements
		float rho0[N[0]] = diagonal elements of the equilibrium ground state density matrix
		float S[2*N[0]] = response function in time (both real and imaginary parts)
*/
__global__ void _evolveS(int* N, float* rU, float* iU, float* d, float* rho0, float *S, int BLOCKDIM){ 
	int t= (gridDim.x*blockIdx.y+ blockIdx.x)*BLOCKDIM*BLOCKDIM+ blockDim.x*threadIdx.y+threadIdx.x;//calculates time of current thread
	if (t>=N[2]) return;//because 16x16 blocks are alocated, some threads at the end are not necessary
	//initialized temporary variables
	float im= 0;
	float re= 0;

	int i, j, a, b;
	for (i=0;i<N[0];i++){
		for (j=0;j<N[0];j++){
			for (a=0;a<N[1];a++){
				for (b=0;b<N[1];b++){
					/*In rU (resp. iU) the relation between rU[n] and indeces i, j, a, b, t is n= i+ N[0]*(a+ N[1]*(j+ N[0]*(b+ N[1]*t))). 
					In d array the relation is n=i+ N[0]* a.*/

					/*real part*/
					re+= d[b*N[0]+j]* d[a*N[0]+i]* rU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]*rho0[i];
					/*imaginary part*/
					im+= d[b*N[0]+j]* d[a*N[0]+i]* iU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]*rho0[i];
				}
			}
		}
	}
	//stores resulsts into S
	S[2*t]= re;
	S[2*t+1]= im;
}

float *nosecuda_polar_1(int* N, float* rU, float* iU, float* d, float* rho0){
	/*creates S array*/
	float *S;
	S= (float*)malloc(2*N[2]*sizeof(float));

	/*initializes device variables*/
	float *_rU, *_iU, *_d, *_rho0, *_S;
	int *_N;
	/*allocate N array*/
	hipMalloc((void**) &_N, 3*sizeof(int));/*length of N is 3*/
	hipMemcpy(_N, N, 3*sizeof(float), hipMemcpyHostToDevice);
	/*allocate rU array*/
	hipMalloc((void**) &_rU, N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float)); 
	hipMemcpy(_rU, rU, N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float), hipMemcpyHostToDevice);
	/*allocate iU array*/
	hipMalloc((void**) &_iU, N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float)); 
	hipMemcpy(_iU, iU, N[0]*N[1]*N[0]*N[1]*N[2]*sizeof(float), hipMemcpyHostToDevice);
	/*allocate d array*/
	hipMalloc((void**) &_d, N[0]*N[1]*sizeof(float));
	hipMemcpy(_d, d, N[0]*N[1]*sizeof(float), hipMemcpyHostToDevice);
	/*allocate rho0 array*/
	hipMalloc((void**) &_rho0, N[0]*sizeof(float));
	hipMemcpy(_rho0, rho0, N[0]*sizeof(float), hipMemcpyHostToDevice);
	/*allocate S array*/
	hipMalloc((void**) &_S, 2*N[2]*sizeof(float));//2*N[2] contains real and imaginarypart

	//calculates grid dimension
	dim3 dimBlock(BLOCKDIM, BLOCKDIM);

	int b= N[2]/(BLOCKDIM*BLOCKDIM)+1;//number of blocks
	int i=1;//grid dimension in blocks
	while ((i+1)*(i+1)<b) i+= 1;//find greatest square grid whose area is lesser than b
	dim3 dimGrid(i, (b-i*i)/i+ 1);

	/*kernel execution*/
	_evolveS<<<dimGrid, dimBlock>>>(_N, _rU, _iU, _d, _rho0, _S, BLOCKDIM);

	/*copy result from the device to the host*/
	hipMemcpy(S, _S, N[2]*2*sizeof(float), hipMemcpyDeviceToHost);

	/*free used memory*/
	hipFree(_N);
	hipFree(_rU);
	hipFree(_iU);
	hipFree(_d);
	hipFree(_rho0);
	hipFree(_S);
	
	/*returns S(t)*/
	return S;//the function which calls this method should free S
}
/*
int main(){//tests nosecuda_polar1 with simple parameters
	float *S;
	//initialization of some parameters / these parameters should make S[0]=1+0i
	int N[3]={2,2,1};
	float rU[16]={1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
	float iU[16]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	float d[4]={1,1,1,1};
	float rho0[2]={1,1};

	S= nosecuda_polar_1(N, rU, iU, d, rho0);//computes S
	int i;
	for (i=0;i<1;i++)//writes S array
		printf("S(%i)=%f+i%f\n", i, S[2*i], S[2*i+1]);
	free(S);
}*/
