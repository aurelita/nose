/*
This is a very simple comparision of nosecuda_polar1.cu and nosecpu_polar1.cu. 
USAGE: 
1) compile it with nvcc:

nvcc -lm nosecuda_polar1.cu nosecpu_polar1.cu nosetest_polar1.cu -o nosetest_polar1
2) run it with an arbitrary trigger parameter. Trigger means that if the real or the imaginary part of S returned from cpu and gpu method differ for more than trigger a new line is printed.

./nosetest_polar1 [trigger]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

extern float *nosecuda_polar_1(int* N, float *rU, float *iU, float *d, float *rho0);
extern float *nosecpu_polar_1(int* N, float *rU, float *iU, float *d, float *rho0);

float random01(){//returns random number between 0 and 1
	return rand()/(float)RAND_MAX;
}

int main(int argc, char **argv){
	//set dimensios of parameters
	int N[3]={4, 4, 300}; 
	float d[N[1]*N[0]]; 
	float rho0[N[0]]; 
	float rU[N[0]*N[1]*N[0]*N[1]*N[2]]; 
	float iU[N[0]*N[1]*N[0]*N[1]*N[2]]; 
	int i, j, a, b, t;

	//set parameters to random values
	for (i=0;i<N[0];i++){//set rho0, d
		rho0[i]= random01()*10.0;
		for (a=0;a<N[1];a++){
			d[i+N[0]*a]= random01()*10;//float between 0 and 100
		}
	}
	for (t=0;t<N[2];t++){//set rU, iU
		for (i=0;i<N[0];i++){//set d
			for (a=0;a<N[1];a++){
				for (j=0;j<N[0];j++){
					for (b=0;b<N[1];b++){
						rU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]= random01();
						iU[i+N[0]*(a+N[1]*(j+N[0]*(b+N[1]*t)))]= random01();
					}
				}
			}
		}
	}
	//now everything is set already and can be used
	printf("starting comparision\n");

	float trigger= 0.0;
	if (argc>1) trigger= atof(argv[1]);
	printf("trigger= %f\n", trigger);

	//execute both cpu and gpu methods
	float *SC, *SG;
	printf("executing cpu method\n");
	SC= (float*)nosecpu_polar_1(N, rU, iU, d, rho0);
	printf("executing gpu method\n");
	SG= (float*)nosecuda_polar_1(N, rU, iU, d, rho0);
	printf("comparing methods\n");
	
	//compare all values and print if differ for more than trigger
	for (t=0;t<N[2];t++){
		if ((fabs(SG[2*t]-SC[2*t])>=trigger)||(fabs(SG[2*t+1]-SC[2*t+1])>= trigger)) printf("for t=%i: SG[t]=%f+i%f, SC[t]=%f+i%f\n", t, SG[2*t], SG[2*t+1], SC[2*t], SC[2*t+1]);
	}
	
	//free memory
	free(SC);
	free(SG);

}
